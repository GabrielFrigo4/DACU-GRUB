#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

// Maneira simples da mat_mul_device
__global__ void mat_mul_device(float* C, const float* A, const float* B, const int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < N && col < N) {
		float sum = 0.0f;
		for (int k = 0; k < N; ++k) {
			sum += A[row * N + k] * B[k * N + col];
		}
		C[row * N + col] = sum;
	}
}

// Maneira simples da mat_mul_host
template <int BLOCK_SIZE>
void mat_mul_host(float* h_C, const float* h_A, const float* h_B, const int N) {
	// Alocar matrizes no device (GPU)
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A, N * N * sizeof(float));
	hipMalloc(&d_B, N * N * sizeof(float));
	hipMalloc(&d_C, N * N * sizeof(float));

	// Copiar dados para o device
	hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

	// Definir tamanho de bloco e grid
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

	// Lançar kernel
	mat_mul_device<<<numBlocks, threadsPerBlock>>>(d_C, d_A, d_B, N);
	hipDeviceSynchronize();

	// Copiar resultado de volta
	hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

	// Liberar memória
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

int main() {
	// Ler entrada N
	int N;
	std::cin >> N;

	// Alocar matrizes no host (CPU)
	float* h_A = new float[N * N];
	float* h_B = new float[N * N];
	float* h_C = new float[N * N];

	// Ler entrada matriz A e matriz B
	for (int i = 0; i < N * N; ++i)
		std::cin >> h_A[i];
	for (int i = 0; i < N * N; ++i)
		std::cin >> h_B[i];

	// Calcular a multiplicação
	mat_mul_host<32>(h_C, h_A, h_B, N);

	// Imprimir resultado
	std::cout << std::fixed << std::setprecision(2);
	for (int y = 0; y < N; ++y) {
		for (int x = 0; x < N; ++x) {
			std::cout << h_C[y * N + x] << " ";
		}
		std::cout << std::endl;
	}

	// Liberar memória
	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	return 0;
}